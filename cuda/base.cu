#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1) / b;
}

static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}


__global__ void mykernel(float* r, const float* d, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= n || j >= n) return;
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k) {
        float x = d[n * i + k];
        float y = d[n * k + j];
        float z = x + y;
        v = min(v, z);
    }
    r[n * i + j] = v;
}


void step(float* r, const float* d, int n) {
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, n * n * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, n * n * sizeof(float)));
    CHECK(hipMemcpy(dGPU, d, n * n * sizeof(float), hipMemcpyHostToDevice));

    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(r, rGPU, n * n * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}
